/*
* This program takes in input a text and a string, being two arrays of char values, and 
* computes how many times the string appears in the text. In particular:
* - function findString receives in input the text and the string and saves in each 
*   position i of a third vector called match, 1 if an occurrence of the string has been 
*   found in the text starting the index i, 0 otherwise.
* - function countMatches receives the vector match in input and count the number of values 
*   equal to 1 (i.e., it counts the number of occurrences of the string in the text).
* - the main function receives as arguments the size of text and string and (for the sake 
*   of brevity) generates randomly the content of the two vectors, invokes the two 
*   functions above and prints the result on the screen.
*/

#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 32
#define MAXVAL 2

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

void printText(char *text, int num);
void findString(char* text, int textDim, char* str, int strDim, char* match);
int countMatches(char *match, int num);
__global__ void findStringKernel(char* text, int textDim, char* str, int strDim, char* match);
__global__ void countMatchesKernel(char *match, int num, int res);
__global__ void findStringKernelShared(char* text, int textDim, char* str, int strDim, char* match);

__global__ void findStringKernel(char* text, int textDim, char* str, int strDim, char* match) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j, ok;
  if (i < (textDim-strDim+1)) {
    for(j=0, ok=1; j<strDim && ok; j++){
      if(text[i+j]!=str[j])
        ok=0;
    }
    match[i] = ok;
  }
}

__global__ void findStringKernelShared(char* text, int textDim, char* str, int strDim, char* match) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  extern __shared__ char tile[];
  int j, ok;

  for (j = threadIdx.x; j < BLOCKSIZE+strDim-1; j+=BLOCKSIZE)
    tile[j] = text[blockIdx.x * blockDim.x + j];
  __syncthreads();

  if (i < (textDim-strDim+1)) {
    for(j=0, ok=1; j<strDim && ok; j++){
      if(tile[i+j]!=str[j])
        ok=0;
    }
    match[i] = ok;
  }
}

__global__ void countMatchesKernel(char *match, int num, int *res) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < num){
    atomicAdd(res, match[i]);
  }
}

//display a vector of numbers on the screen
void printText(char *text, int num) {
  int i;
  for(i=0; i<num; i++)
    printf("%c", text[i]);
  printf("\n");    
}

//kernel function 1: identify strings in the text
void findString(char* text, int textDim, char* str, int strDim, char* match) {
  int i, j, ok;
  for(i=0; i<textDim-strDim+1; i++){
    for(j=0, ok=1; j<strDim && ok; j++){
      if(text[i+j]!=str[j])
        ok=0;
    }
    match[i] = ok;
  }
}

//kernel function 2: count matches
int countMatches(char *match, int num) {
  int i, count;
  for(i=0, count=0; i<num; i++)
    count+=match[i];
  return count;    
}

int main(int argc, char **argv) {
  char *text, *str, *match;
  int count;
  int textDim, strDim;
  int i;

  //read arguments
  if(argc!=3){
    printf("Please specify sizes of the two input vectors\n");
    return 0;
  }
  textDim=atoi(argv[1]);
  strDim=atoi(argv[2]);
  
  //allocate memory for the three vectors
  text = (char*) malloc(sizeof(char) * (textDim));
  if(!text){
    printf("Error: malloc failed\n");
    return 1;
  }
  str = (char*) malloc(sizeof(char) * (strDim));
  if(!str){
    printf("Error: malloc failed\n");
    return 1;
  }
  match = (char*) malloc(sizeof(char) * (textDim-strDim+1));
  if(!match){
    printf("Error: malloc failed\n");
    return 1;
  }

  //initialize input vectors
  srand(0);
  for(i=0; i<textDim; i++)
    text[i] = rand()%MAXVAL +'a';

  for(i=0; i<strDim; i++)
    str[i] = rand()%MAXVAL +'a';
  
  char *text_d, *str_d, *match_d;
  int *res;
  CHECK(hipMalloc((void**) &res, sizeof(int)));
  CHECK(hipMalloc((void**) &text_d, sizeof(char) * textDim));
  CHECK(hipMalloc((void**) &str_d, sizeof(char) * strDim));
  CHECK(hipMalloc((void**) &match_d, sizeof(char) * (textDim-strDim+1)));

  int *(count_d);
  *count_d = 0;
  CHECK(hipMemcpy(res, count_d, sizeof(int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(text_d, text, sizeof(char) * textDim, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(str_d, str, sizeof(char) * strDim, hipMemcpyHostToDevice));

  dim3 blocksPerGrid((textDim - strDim) / BLOCKSIZE +1);
  dim3 threadsPerBlock(BLOCKSIZE);

  findStringKernel<<<blocksPerGrid, threadsPerBlock>>>(text_d, textDim, str_d, strDim, match_d);
  CHECK_KERNELCALL();
  countMatchesKernel<<<blocksPerGrid, threadsPerBlock>>>(match_d, textDim-strDim+1, res);
  CHECK_KERNELCALL();
  CHECK(hipMemcpy(count_d, res, sizeof(int), hipMemcpyDeviceToHost));

  findStringKernelShared<<<blocksPerGrid, threadsPerBlock, (BLOCKSIZE + strDim) * sizeof(char)>>>(text_d, textDim, str_d, strDim, match_d);
  CHECK_KERNELCALL();
  countMatchesKernel<<<blocksPerGrid, threadsPerBlock>>>(match_d, textDim-strDim+1, res);
  CHECK_KERNELCALL();
  CHECK(hipMemcpy(count_d, res, sizeof(int), hipMemcpyDeviceToHost));

  CHECK(hipFree(res));
  CHECK(hipFree(text_d));
  CHECK(hipFree(str_d));
  CHECK(hipFree(match_d));
  
  //execute on CPU  
  findString(text, textDim, str, strDim, match);
  count = countMatches(match, textDim-strDim+1);
 
  //print results
  printText(text, textDim);
  printText(str, strDim);
  printf("%d\n", count);
  
  free(text);
  free(str);
  free(match);
  
  return 0;
}
