#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define DIM           (1024)
#define BLOCK_DIM     DIM
#define STRIDE_FACTOR 2

// CPU version of the scan kernel EXCLUSIVE
void scan_cpu(const float* input, float* output, const int length) {
  output[0] = 0;
  for (int i = 1; i < length; ++i) { output[i] = output[i - 1] + input[i - 1]; }
}

// GPU version of the scan kernel EXCLUSIVE
__global__ void
    naive_scan_gpu(const float* __restrict__ input, float* __restrict__ output, const int length) {
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  float sum{0};

  for (unsigned int i = 0; i < index; i++) { sum += input[i]; }
  
  output[index] = sum;

  return;
}

int main() {
  std::vector<float> input(DIM);
  std::vector<float> output(DIM);
  for (int i = 0; i < DIM; ++i)
    input[i] = static_cast<float>(rand()) / RAND_MAX; // Random value between 0 and 1 }

  // CPU version
  scan_cpu(input.data(), output.data(), input.size());

  // GPU version
  float *input_d, *output_d;
  CHECK(hipMalloc(&input_d, DIM * sizeof(float)));
  CHECK(hipMalloc(&output_d, DIM * sizeof(float)));
  CHECK(hipMemcpy(input_d, input.data(), DIM * sizeof(float), hipMemcpyHostToDevice));

  const dim3 block_dim(BLOCK_DIM, 1, 1);
  const dim3 grid_dim((DIM + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
  naive_scan_gpu<<<grid_dim, block_dim>>>(input_d, output_d, DIM);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  std::vector<float> host_result(DIM);
  CHECK(hipMemcpy(host_result.data(), output_d, DIM * sizeof(float), hipMemcpyDeviceToHost));

  for (int i = 0; i < host_result.size(); ++i) {
    if (std::abs(output[i] - host_result[i]) > 1e-3) {
      std::cout << "Scan CPU and GPU are NOT equivalent!" << std::endl;
      std::cout << "Index: " << i << std::endl;
      std::cout << "CPU: " << output[i] << std::endl;
      std::cout << "GPU: " << host_result[i] << std::endl;
      return EXIT_FAILURE;
    }
  }

  std::cout << "Scan CPU and GPU are equivalent!" << std::endl;

  // Cleanup
  CHECK(hipFree(output_d));
  CHECK(hipFree(input_d));

  return 0;
}
