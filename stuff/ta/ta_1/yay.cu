#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define CHECK(call)                                               \
    {                                                             \
        hipError_t checkErr = (call);                            \
        if (checkErr != hipSuccess)                              \
        {                                                         \
            printf("Failed: %s\n", hipGetErrorString(checkErr)); \
            assert(checkErr == hipSuccess);                      \
        }                                                         \
    }

#define CHECK_LAST() CHECK(hipGetLastError())

__global__ void saxpy(int const n,
                      float const a,
                      int const *const __restrict__ x,
                      int *const __restrict__ y)
{
    int const tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
        y[tid] = a * x[tid] + y[tid];
}

int main(int const argc, char **argv)
{
    int *x_h, *y_h;
    int *x_d, *y_d;

    constexpr int N = 1 << 20;
    const int blockSize = 256;
    const float multiplier = .14f;

    CHECK(hipMalloc(&x_d, N * sizeof(*x_d)));
    CHECK(hipMalloc(&y_d, N * sizeof(*y_d)));

    CHECK(hipHostMalloc(&x_h, N * sizeof(*x_h), hipHostMallocDefault));
    CHECK(hipHostMalloc(&y_h, N * sizeof(*y_h), hipHostMallocDefault));

    for (int i = 0; i < N; ++i)
    {
        x_h[i] = i;
        y_h[i] = N - i;
    }

    CHECK(hipMemcpy(x_d, x_h, N * sizeof(*x_h), hipMemcpyHostToDevice));

    saxpy<<<(N + blockSize - 1) / blockSize, blockSize>>>(N, multiplier, x_d, y_d);
    CHECK_LAST();

    CHECK(hipMemcpy(y_h, y_d, N * sizeof(*y_d), hipMemcpyDeviceToHost));

    hipHostFree(x_h);
    hipHostFree(y_h);
    hipFree(x_d);
    hipFree(y_d);
    return 0;
}