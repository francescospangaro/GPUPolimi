
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <vector>

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define SIZE_A 20000
#define SIZE_B 12768
#define SIZE_C (SIZE_A + SIZE_B)

#define BLOCK_DIM 128
#define GRID_DIM 32
#define TILE_SIZE 1024

__device__ __forceinline__ int gpu_ceil(const int a, const int b) { return ceil((double)a / b); }
__device__ __forceinline__ int gpu_min(const int a, const int b) { return min(a, b); }
__device__ __forceinline__ int gpu_max(const int a, const int b) { return max(a, b); }

// Function for CPU and GPU merge sequentia computation
__device__ __host__ void
merge_sequential(const int *A, const int dim_A, const int *B, const int dim_B, int *C)
{
  int i = 0;
  int j = 0;
  int k = 0;

  while ((i < dim_A) && (j < dim_B))
  {
    if (A[i] <= B[j])
      C[k++] = A[i++];
    else
      C[k++] = B[j++];
  }
  if (i == dim_A)
  {
    while (j < dim_B)
    {
      C[k++] = B[j++];
    }
  }
  else
  {
    while (i < dim_A)
    {
      C[k++] = A[i++];
    }
  }
}

__device__ void merge_sequential_circular(const int *__restrict__ A,
                                          const int m,
                                          const int *__restrict__ B,
                                          const int n,
                                          int *__restrict__ C,
                                          const int A_S_start,
                                          const int B_S_start,
                                          const int tile_size)
{
  int i = 0; // Virtual index into A
  int j = 0; // Virtual index into B
  int k = 0; // virtual index into C

  while ((i < m) && (j < n))
  {
    int i_cir = (A_S_start + i) % tile_size;
    int j_cir = (B_S_start + j) % tile_size;

    // TODO
    if (A[i_cir] <= B[j_cir])
    {
      C[k++] = A[i_cir];
      i++;
    }
    else
    {
      C[k++] = B[j_cir];
      j++;
    }
  }

  if (i == m)
  { // Done with A[] handle remaining B[]
    // TODO
    for (; j < n; ++j)
      C[k++] = B[(B_S_start + j) % tile_size];
  }
  else
  { // Done with B[], handle remaining A[]
    // TODO
    for (; i < m; ++i)
      C[k++] = A[(A_S_start + i) % tile_size];
  }
}

// Function to compute the co-rank
__device__ int
co_rank(const int k, const int *__restrict__ A, const int m, const int *__restrict__ B, const int n)
{
  int i = gpu_min(k, m); // min
  int j = k - i;
  int i_low = gpu_max(0, (k - n)); // max
  int j_low = gpu_max(0, (k - m));
  int delta;
  bool active = true;
  while (active)
  {
    // TODO checks for binary search of the co-rank
    if (i > 0 && j < n && A[i - 1] > B[j])
    {
      delta = (i - i_low + 1) / 2;
      j_low = j;
      j = j + delta;
      i = i - delta;
      continue;
    }

    if (j > 0 && i < m && B[j - 1] >= A[i])
    {
      delta = (j - j_low + 1) / 2;
      i_low = i;
      i = i + delta;
      j = j - delta;
      continue;
    }

    active = false;
  }
  return i;
}

// Function to compute the co-rank circular version
__device__ int co_rank_circular(const int k,
                                const int *__restrict__ A,
                                const int m,
                                const int *__restrict__ B,
                                const int n,
                                const int A_S_start,
                                const int B_S_start,
                                const int tile_size)
{
  int i = gpu_min(k, m); // min
  int j = k - i;
  int i_low = gpu_max(0, (k - n)); // max
  int j_low = gpu_max(0, (k - m));
  int delta;
  bool active = true;
  while (active)
  {
    // TODO checks for binary search of the co-rank
    const int i_cir = (A_S_start + i) % tile_size;
    const int i_m_1_cir = (A_S_start + i - 1) % tile_size;
    const int j_cir = (B_S_start + j) % tile_size;
    const int j_m_1_cir = (B_S_start + j - 1) % tile_size;

    if (i > 0 && j < n && A[i_m_1_cir] > B[j_cir])
    {
      delta = (i - i_low + 1) / 2;
      j_low = j;
      j = j + delta;
      i = i - delta;
      continue;
    }

    if (j > 0 && i < m && B[j_m_1_cir] >= A[i_cir])
    {
      delta = (j - j_low + 1) / 2;
      i_low = i;
      i = i + delta;
      j = j - delta;
      continue;
    }

    active = false;
  }
  return i;
}

// Function for GPU merge computation
__global__ void merge_tiled_gpu(const int *__restrict__ A,
                                const int dim_A,
                                const int *__restrict__ B,
                                const int dim_B,
                                int *__restrict__ C)
{
  __shared__ int A_s[TILE_SIZE];
  __shared__ int B_s[TILE_SIZE];

  const int dim_C = dim_A + dim_B;

  const int C_curr = blockIdx.x * gpu_ceil(dim_C, gridDim.x);
  const int C_next = gpu_min((blockIdx.x + 1) * gpu_ceil(dim_C, gridDim.x), dim_C);

  if (threadIdx.x == 0)
  {
    A_s[0] = co_rank(C_curr, A, dim_A, B, dim_B);
    A_s[1] = co_rank(C_next, A, dim_A, B, dim_B);
  }
  __syncthreads();

  const int A_curr = A_s[0];
  const int A_next = A_s[1];
  const int B_curr = C_curr - A_curr;
  const int B_next = C_next - A_next;
  __syncthreads();

  int counter = 0;
  const int C_length = C_next - C_curr;
  const int A_length = A_next - A_curr;
  const int B_length = B_next - B_curr;
  const int total_iteration = gpu_ceil(C_length, TILE_SIZE);
  int C_completed = 0;
  int A_consumed = 0;
  int B_consumed = 0;
  int A_read = 0;
  int B_read = 0;
  int A_S_start = 0;
  int B_S_start = 0;
  int A_S_consumed = TILE_SIZE; // In the first iteration, fill the TILE_SIZE
  int B_S_consumed = TILE_SIZE; // In the first iteration, fill the tile_size

  while (counter < total_iteration)
  {
    // Portion of the computation remaining to be done
    const int C_remaining = C_length - C_completed;
    const int C_remaining_tile = gpu_min(TILE_SIZE, C_remaining);
    const int A_remaining = A_length - A_consumed;
    const int A_remaining_tile = gpu_min(TILE_SIZE, A_remaining);
    const int B_remaining = B_length - B_consumed;
    const int B_remaining_tile = gpu_min(TILE_SIZE, B_remaining);

    // Loading (refilling) A_S_consumed elements into A_S
    // TODO
    for (int i = threadIdx.x; i < A_S_consumed; i += blockDim.x)
    {
      if (i < A_remaining)
        A_s[(A_S_start + (TILE_SIZE - A_S_consumed) + i) % TILE_SIZE] = A[A_curr + i + A_read];
    }
    A_read += gpu_min(A_remaining, A_S_consumed);

    // Loading B_S_consumed elements into B_S
    // TODO
    for (int i = threadIdx.x; i < B_S_consumed; i += blockDim.x)
    {
      if (i < B_remaining)
        B_s[(B_S_start + (TILE_SIZE - B_S_consumed) + i) % TILE_SIZE] = B[B_curr + i + B_read];
    }
    B_read += gpu_min(B_remaining, B_S_consumed);
    __syncthreads();

    // Get sections of A, B, and C
    // TODO
    const size_t k_curr = gpu_min(threadIdx.x * (TILE_SIZE / blockDim.x), C_length - C_completed);
    const size_t k_next = gpu_min((threadIdx.x + 1) * (TILE_SIZE / blockDim.x), C_length - C_completed);

    const size_t i_curr = co_rank_circular(k_curr, A_s, A_remaining_tile, B_s, B_remaining_tile, A_S_start, B_S_start, TILE_SIZE);
    const size_t i_next = co_rank_circular(k_next, A_s, A_remaining_tile, B_s, B_remaining_tile, A_S_start, B_S_start, TILE_SIZE);

    const size_t j_curr = k_curr - i_curr;
    const size_t j_next = k_next - i_next;

    // Do merge in parallel
    // TODO
    merge_sequential_circular(A_s, i_next - i_curr,
                              B_s, j_next - j_curr,
                              &C[C_curr + C_completed + k_curr],
                              A_S_start + i_curr,
                              B_S_start + j_curr,
                              TILE_SIZE);

    // Update counter and number of computed elements
    // TODO
    counter++;
    A_S_consumed = co_rank_circular(C_remaining_tile, A_s, A_remaining_tile, B_s, B_remaining_tile, A_S_start, B_S_start, TILE_SIZE);
    B_S_consumed = C_remaining_tile - A_S_consumed;
    A_consumed += A_S_consumed;
    C_completed += C_remaining_tile;
    B_consumed = C_completed - A_consumed;

    A_S_start = A_S_start + A_S_consumed;
    if (A_S_start >= TILE_SIZE)
      A_S_start = A_S_start - TILE_SIZE;

    B_S_start = B_S_start + B_S_consumed;
    if (B_S_start >= TILE_SIZE)
      B_S_start = B_S_start - TILE_SIZE;

    __syncthreads();
  }
}

// simple sort, just to sort the randomly generated arrays
void sort_array(int *arr, const int dim)
{
  int i, j;

  for (i = 0; i < dim; i++)
  {
    for (j = i + 1; j < dim; j++)
    {
      if (arr[j] < arr[i])
      {
        int tmp = arr[i];
        arr[i] = arr[j];
        arr[j] = tmp;
      }
    }
  }
}

int main()
{
  static_assert((SIZE_C / GRID_DIM) % TILE_SIZE == 0, "Total size should be a multiple of TILE_SIZE");
  // Generate random input data
  std::vector<int> A_host(SIZE_A);
  std::vector<int> B_host(SIZE_B);
  for (int i = 0; i < SIZE_A; ++i)
  {
    A_host[i] = rand() % 1000;
  }
  for (int i = 0; i < SIZE_B; ++i)
  {
    B_host[i] = rand() % 1000;
  }
  sort_array(A_host.data(), SIZE_A);
  sort_array(B_host.data(), SIZE_B);

  // Compute merge on CPU
  std::vector<int> cpu_result(SIZE_C);
  merge_sequential(A_host.data(), SIZE_A, B_host.data(), SIZE_B, cpu_result.data());

  // Allocate memory on GPU
  int *A_device, *B_device, *C_device;
  CHECK(hipMalloc(&A_device, SIZE_A * sizeof(int)));
  CHECK(hipMalloc(&B_device, SIZE_B * sizeof(int)));
  CHECK(hipMalloc(&C_device, SIZE_C * sizeof(int)));
  CHECK(hipMemcpy(A_device, A_host.data(), SIZE_A * sizeof(int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_device, B_host.data(), SIZE_B * sizeof(int), hipMemcpyHostToDevice));

  // Configure GPU kernel launch
  dim3 threads_per_block(BLOCK_DIM);
  dim3 blocks_per_grid(GRID_DIM);
  // Launch GPU kernel
  merge_tiled_gpu<<<blocks_per_grid, threads_per_block>>>(A_device, SIZE_A, B_device, SIZE_B, C_device);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  // Copy result back to host
  std::vector<int> gpu_result(SIZE_C);
  CHECK(hipMemcpy(gpu_result.data(), C_device, SIZE_C * sizeof(int), hipMemcpyDeviceToHost));

  // Compare CPU and GPU results
  for (int i = 0; i < SIZE_C; ++i)
    if (cpu_result[i] != gpu_result[i])
    {
      std::cout << "Merge CPU and GPU are NOT equivalent!" << std::endl;
      std::cout << "Index: " << i << std::endl;
      std::cout << "CPU: " << cpu_result[i] << std::endl;
      std::cout << "GPU: " << gpu_result[i] << std::endl;
      return EXIT_FAILURE;
    }

  std::cout << "Merge CPU and GPU are equivalent!" << std::endl;

  // Free memory
  CHECK(hipFree(A_device));
  CHECK(hipFree(B_device));
  CHECK(hipFree(C_device));

  return EXIT_SUCCESS;
}
