
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <vector>

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define SIZE_A 20000
#define SIZE_B 12768
#define SIZE_C (SIZE_A + SIZE_B)

#define BLOCK_DIM 128
#define GRID_DIM 32
#define TILE_SIZE 1024

__device__ __forceinline__ int gpu_ceil(const int a, const int b) { return ceil((double)a / b); }
__device__ __forceinline__ int gpu_min(const int a, const int b) { return min(a, b); }
__device__ __forceinline__ int gpu_max(const int a, const int b) { return max(a, b); }

// Function for CPU and GPU merge sequentia computation
__device__ __host__ void
merge_sequential(const int *A, const int dim_A, const int *B, const int dim_B, int *C)
{
  int i = 0;
  int j = 0;
  int k = 0;

  while ((i < dim_A) && (j < dim_B))
  {
    if (A[i] <= B[j])
      C[k++] = A[i++];
    else
      C[k++] = B[j++];
  }
  if (i == dim_A)
  {
    while (j < dim_B)
    {
      C[k++] = B[j++];
    }
  }
  else
  {
    while (i < dim_A)
    {
      C[k++] = A[i++];
    }
  }
}

// Function to compute the co-rank
__device__ int
co_rank(const int k, const int *__restrict__ A, const int m, const int *__restrict__ B, const int n)
{
  int i = gpu_min(k, m); // min
  int j = k - i;
  int i_low = gpu_max(0, (k - n)); // max
  int j_low = gpu_max(0, (k - m));
  int delta;
  bool active = true;
  while (active)
  {
    // TODO checks for binary search of the co-rank
    if (i > 0 && j < n && A[i - 1] > B[j])
    {
      delta = (i - i_low + 1) / 2;
      j_low = j;
      j = j + delta;
      i = i - delta;
      continue;
    }

    if (j > 0 && i < m && B[j - 1] >= A[i])
    {
      delta = (j - j_low + 1) / 2;
      i_low = i;
      i = i + delta;
      j = j - delta;
      continue;
    }

    active = false;
  }
  return i;
}

// Function for GPU merge computation
__global__ void merge_tiled_gpu(const int *__restrict__ A,
                                const int dim_A,
                                const int *__restrict__ B,
                                const int dim_B,
                                int *__restrict__ C)
{
  __shared__ int A_s[TILE_SIZE];
  __shared__ int B_s[TILE_SIZE];

  const int dim_C = dim_A + dim_B;

  const int C_curr = blockIdx.x * gpu_ceil(dim_C, gridDim.x);
  const int C_next = gpu_min((blockIdx.x + 1) * gpu_ceil(dim_C, gridDim.x), dim_C);

  if (threadIdx.x == 0)
  {
    A_s[0] = co_rank(C_curr, A, dim_A, B, dim_B);
    A_s[1] = co_rank(C_next, A, dim_A, B, dim_B);
  }
  __syncthreads();

  const int A_curr = A_s[0];
  const int A_next = A_s[1];
  const int B_curr = C_curr - A_curr;
  const int B_next = C_next - A_next;

  int counter = 0;
  const int C_length = C_next - C_curr;
  const int A_length = A_next - A_curr;
  const int B_length = B_next - B_curr;
  const int total_iteration = gpu_ceil(C_length, TILE_SIZE);
  int C_completed = 0;
  int A_consumed = 0;
  int B_consumed = 0;
  while (counter < total_iteration)
  {
    // Loading tile into shared memory
    // TODO
    for (size_t i = threadIdx.x; i < TILE_SIZE; i += blockDim.x)
    {
      if (i < A_length - A_consumed)
        A_s[i] = A[A_curr + A_consumed + i];
    }
    for (size_t i = threadIdx.x; i < TILE_SIZE; i += blockDim.x)
    {
      if (i < B_length - B_consumed)
        B_s[i] = B[B_curr + B_consumed + i];
    }
    __syncthreads();

    // Get sections of A, B, and C
    // TODO
    const size_t k_curr = gpu_min(threadIdx.x * (TILE_SIZE / blockDim.x), C_length - C_completed);
    const size_t k_next = gpu_min((threadIdx.x + 1) * (TILE_SIZE / blockDim.x), C_length - C_completed);

    const size_t i_curr = co_rank(k_curr, A_s, A_length, B_s, B_length);
    const size_t i_next = co_rank(k_next, A_s, A_length, B_s, B_length);

    const size_t j_curr = k_curr - i_curr;
    const size_t j_next = k_next - i_next;

    // Merge and update counter and completed elements
    // TODO
    merge_sequential(&A_s[i_curr], i_next - i_curr, &B_s[j_curr], j_next - j_curr, &C[C_curr + C_completed + k_curr]);

    // Update counter and number of computed elements
    // TODO
    counter++;
    C_completed += TILE_SIZE;
    A_consumed += i_next;
    B_consumed = C_completed - A_consumed;
    __syncthreads();
  }
}

// simple sort, just to sort the randomly generated arrays
void sort_array(int *arr, const int dim)
{
  int i, j;

  for (i = 0; i < dim; i++)
  {
    for (j = i + 1; j < dim; j++)
    {
      if (arr[j] < arr[i])
      {
        int tmp = arr[i];
        arr[i] = arr[j];
        arr[j] = tmp;
      }
    }
  }
}

int main()
{
  static_assert((SIZE_C / GRID_DIM) % TILE_SIZE == 0, "Total size should be a multiple of TILE_SIZE");
  // Generate random input data
  std::vector<int> A_host(SIZE_A);
  std::vector<int> B_host(SIZE_B);
  for (int i = 0; i < SIZE_A; ++i)
  {
    A_host[i] = rand() % 1000;
  }
  for (int i = 0; i < SIZE_B; ++i)
  {
    B_host[i] = rand() % 1000;
  }
  sort_array(A_host.data(), SIZE_A);
  sort_array(B_host.data(), SIZE_B);

  // Compute merge on CPU
  std::vector<int> cpu_result(SIZE_C);
  merge_sequential(A_host.data(), SIZE_A, B_host.data(), SIZE_B, cpu_result.data());

  // Allocate memory on GPU
  int *A_device, *B_device, *C_device;
  CHECK(hipMalloc(&A_device, SIZE_A * sizeof(int)));
  CHECK(hipMalloc(&B_device, SIZE_B * sizeof(int)));
  CHECK(hipMalloc(&C_device, SIZE_C * sizeof(int)));
  CHECK(hipMemcpy(A_device, A_host.data(), SIZE_A * sizeof(int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_device, B_host.data(), SIZE_B * sizeof(int), hipMemcpyHostToDevice));

  // Configure GPU kernel launch
  dim3 threads_per_block(BLOCK_DIM);
  dim3 blocks_per_grid(GRID_DIM);
  // Launch GPU kernel
  merge_tiled_gpu<<<blocks_per_grid, threads_per_block>>>(A_device, SIZE_A, B_device, SIZE_B, C_device);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  // Copy result back to host
  std::vector<int> gpu_result(SIZE_C);
  CHECK(hipMemcpy(gpu_result.data(), C_device, SIZE_C * sizeof(int), hipMemcpyDeviceToHost));

  // Compare CPU and GPU results
  for (int i = 0; i < SIZE_C; ++i)
    if (cpu_result[i] != gpu_result[i])
    {
      std::cout << "Merge CPU and GPU are NOT equivalent!" << std::endl;
      std::cout << "Index: " << i << std::endl;
      std::cout << "CPU: " << cpu_result[i] << std::endl;
      std::cout << "GPU: " << gpu_result[i] << std::endl;
      return EXIT_FAILURE;
    }

  std::cout << "Merge CPU and GPU are equivalent!" << std::endl;

  // Free memory
  CHECK(hipFree(A_device));
  CHECK(hipFree(B_device));
  CHECK(hipFree(C_device));

  return EXIT_SUCCESS;
}
