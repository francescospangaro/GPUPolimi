
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define BLOCK_DIM 8
#define IN_TILE_DIM BLOCK_DIM
#define OUT_TILE_DIM (IN_TILE_DIM - 2)

#define Z_SLICING 32

// #define DIM (512 + 256)
#define DIM (96)
#define SIZE (DIM * DIM * DIM)

#define get(data, i, j, k, N) data[(i) * N * N + (j) * N + (k)]

#define c0 1
#define c1 1
#define c2 1
#define c3 1
#define c4 1
#define c5 1
#define c6 1

// Kernel function for GPU
__global__ void
stencil_kernel_register_tiling_gpu(const float *__restrict__ in, float *__restrict__ out, const int N)
{
  const unsigned int i_start = blockIdx.z * Z_SLICING;
  const unsigned int j = blockIdx.y * OUT_TILE_DIM + threadIdx.y - 1;
  const unsigned int k = blockIdx.x * OUT_TILE_DIM + threadIdx.x - 1;

  float in_prev;
  __shared__ int in_curr_s[IN_TILE_DIM][IN_TILE_DIM];
  float in_next;
  // Check greater than not needed since index is an unsigned int
  // TODO load shared memory
  if (i_start - 1 < N && j < N && k < N)
    in_prev = get(in, i_start - 1, j, k, N);
  if (i_start < N && j < N && k < N)
    in_curr_s[threadIdx.y][threadIdx.x] = get(in, i_start, j, k, N);

  for (unsigned int i = i_start; i < i_start + Z_SLICING; ++i)
  {
    // Check greater than not needed since index is an unsigned int
    // TODO load shared memory
    if (i + 1 < N && j < N && k < N)
      in_next = get(in, i + 1, j, k, N);
    __syncthreads();

    // TODO perform stencil computation
    if (i >= 1 && i < N - 1 &&
        j >= 1 && j < N - 1 &&
        k >= 1 && k < N - 1 &&
        threadIdx.x >= 1 && threadIdx.x < IN_TILE_DIM - 1 &&
        threadIdx.y >= 1 && threadIdx.y < IN_TILE_DIM - 1)
    {
      get(out, i, j, k, N) = c0 * in_curr_s[threadIdx.y][threadIdx.x] +
                             c1 * in_curr_s[threadIdx.y][threadIdx.x - 1] +
                             c2 * in_curr_s[threadIdx.y][threadIdx.x + 1] +
                             c3 * in_curr_s[threadIdx.y - 1][threadIdx.x] +
                             c4 * in_curr_s[threadIdx.y + 1][threadIdx.x] +
                             c5 * in_prev +
                             c6 * in_next;
    }
    __syncthreads();

    // TODO switch shared memory
    in_prev = in_curr_s[threadIdx.y][threadIdx.x];
    in_curr_s[threadIdx.y][threadIdx.x] = in_next;
  }
}

// Function for CPU stencil computation
void stencil_cpu(const float *in, float *out, const int N)
{
  for (int i = 1; i < N - 1; ++i)
    for (int j = 1; j < N - 1; ++j)
      for (int k = 1; k < N - 1; ++k)
        get(out, i, j, k, N) = c0 * get(in, i, j, k, N) + c1 * get(in, i, j, k - 1, N) +
                               c2 * get(in, i, j, k + 1, N) + c3 * get(in, i, j - 1, k, N) +
                               c4 * get(in, i, j + 1, k, N) + c5 * get(in, i - 1, j, k, N) +
                               c6 * get(in, i + 1, j, k, N);
}

int main()
{
  static_assert(DIM / Z_SLICING, "The dimension should be divisible by the Z slicing");
  static_assert(DIM / OUT_TILE_DIM, "The dimension should be divisible by the number of output block computer by each block");
  // Generate random input data
  std::vector<float> input_data(SIZE);
  for (int i = 0; i < SIZE; ++i)
  {
    input_data[i] = rand() % 10;
  }

  // Compute stencil on CPU
  std::vector<float> cpu_result(SIZE);
  stencil_cpu(input_data.data(), cpu_result.data(), DIM);

  // Allocate memory on GPU
  float *input_data_gpu, *output_data_gpu;
  CHECK(hipMalloc(&input_data_gpu, SIZE * sizeof(float)));
  CHECK(hipMalloc(&output_data_gpu, SIZE * sizeof(float)));
  CHECK(hipMemcpy(input_data_gpu, input_data.data(), SIZE * sizeof(float), hipMemcpyHostToDevice));

  // Configure GPU kernel launch
  dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
  dim3 blocks_per_grid((DIM + OUT_TILE_DIM - 1) / OUT_TILE_DIM,
                       (DIM + OUT_TILE_DIM - 1) / OUT_TILE_DIM,
                       (DIM + Z_SLICING - 1) / Z_SLICING);

  // Launch GPU kernel
  stencil_kernel_register_tiling_gpu<<<blocks_per_grid, threads_per_block>>>(input_data_gpu,
                                                                             output_data_gpu,
                                                                             DIM);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  // Copy result back to host
  std::vector<float> gpu_result(SIZE);
  CHECK(hipMemcpy(gpu_result.data(), output_data_gpu, SIZE * sizeof(float), hipMemcpyDeviceToHost));

  // Compare CPU and GPU results
  for (int i = 0; i < SIZE; ++i)
    if (cpu_result[i] != gpu_result[i])
    {
      std::cout << "Stencil CPU and GPU are NOT equivalent!" << std::endl;
      std::cout << "Index: " << i << std::endl;
      std::cout << "CPU: " << cpu_result[i] << std::endl;
      std::cout << "GPU: " << gpu_result[i] << std::endl;
      return EXIT_FAILURE;
    }

  std::cout << "Stencil CPU and GPU are equivalent!" << std::endl;

  // Free memory
  CHECK(hipFree(input_data_gpu));
  CHECK(hipFree(output_data_gpu));

  return EXIT_SUCCESS;
}
