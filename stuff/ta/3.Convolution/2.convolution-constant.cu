#include <cassert>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

using input_type = float;
using filter_type = input_type;

#define FILTER_RADIUS 4
#define FILTER_SIZE (FILTER_RADIUS * 2 + 1)
__constant__ filter_type constant_filter[FILTER_SIZE][FILTER_SIZE];

#define IN_TILE_DIM 32

#define THREADS_PER_AXIS IN_TILE_DIM

void convolution_cpu(input_type *input,
                     const input_type *filter,
                     input_type *output,
                     const int width,
                     const int height)
{
  for (int outRow = 0; outRow < width; outRow++)
  {
    for (int outCol = 0; outCol < height; outCol++)
    {
      input_type value{0.0f};
      for (int row = 0; row < FILTER_SIZE; row++)
        for (int col = 0; col < FILTER_SIZE; col++)
        {
          int inRow = outRow - FILTER_RADIUS + row;
          int inCol = outCol - FILTER_RADIUS + col;
          if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width)
          {
            value += filter[row * FILTER_SIZE + col] * input[inRow * width + inCol];
          }
        }
      output[outRow * width + outCol] = value;
    }
  }
}

// GPU filter for convolution CONSTANT MEMORY
__global__ void convolution_constant_mem_kernel(const input_type *__restrict__ input,
                                                input_type *__restrict__ output,
                                                const int width,
                                                const int height)
{
  const int outCol = blockIdx.x * blockDim.x + threadIdx.x;
  const int outRow = blockIdx.y * blockDim.y + threadIdx.y;

  input_type value{0.0f};
#pragma unroll
  for (int row = 0; row < FILTER_SIZE; row++)
#pragma unroll
    for (int col = 0; col < FILTER_SIZE; col++)
    {
      const int inRow = outRow - FILTER_RADIUS + row;
      const int inCol = outCol - FILTER_RADIUS + col;
      if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width)
      {
        value += constant_filter[row][col] * input[inRow * width + inCol];
      }
    }
  output[outRow * width + outCol] = value;
}

int main(int argc, char **argv)
{
  if (argc != 2)
  {
    printf("Please specify matrix dimensions\n");
    return EXIT_FAILURE;
  }
  const unsigned dim = atoi(argv[1]);
  assert(dim % THREADS_PER_AXIS == 0);
  const unsigned int width = dim;
  const unsigned int height = dim;

  input_type *input = new input_type[width * height];               // Input
  filter_type *filter = new filter_type[FILTER_SIZE * FILTER_SIZE]; // Convolution filter
  input_type *output_cpu = new input_type[width * height];          // Output (CPU)
  input_type *output_gpu = new input_type[width * height];          // Output (GPU)

  // Randomly initialize the inputs
  for (int i = 0; i < FILTER_SIZE * FILTER_SIZE; i++)
    filter[i] = static_cast<filter_type>(rand()) / RAND_MAX;

  for (int i = 0; i < width * height; ++i)
    input[i] = static_cast<input_type>(rand()) / RAND_MAX; // Random value between 0 and 1

  // Call CPU convolution
  convolution_cpu(input, filter, output_cpu, width, height);

  // Allocate GPU memory and copy data
  input_type *d_input, *d_output;
  CHECK(hipMalloc(&d_input, width * height * sizeof(input_type)));
  CHECK(hipMalloc(&d_output, width * height * sizeof(input_type)));
  CHECK(hipMemcpy(d_input, input, width * height * sizeof(input_type), hipMemcpyHostToDevice));

  //  Call CONSTANT MEM GPU convolution kernel
  // Same (block, grid) as above
  const dim3 threadsPerBlock(THREADS_PER_AXIS, THREADS_PER_AXIS);
  const dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constant_filter), filter, FILTER_SIZE * FILTER_SIZE * sizeof(filter_type)));
  convolution_constant_mem_kernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width, height);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  // Copy results back to host
  CHECK(hipMemcpy(output_gpu, d_output, width * height * sizeof(input_type), hipMemcpyDeviceToHost));

  // Compare results (e.g., check if output_cpu and output_gpu are equivalent)
  for (int i = 0; i < width * height; i++)
  {
    if (std::abs(output_cpu[i] - output_gpu[i]) > 1e-3)
    {
      std::cout << "Convolution CONSTANT MEMORY results are NOT equivalent!" << std::endl;
      return EXIT_FAILURE;
    }
  }

  std::cout << "Convolution CONSTANT MEMORY results are equivalent!" << std::endl;

  // Cleanup and deallocate memory
  delete[] input;
  delete[] filter;
  delete[] output_cpu;
  delete[] output_gpu;
  CHECK(hipFree(d_input));
  CHECK(hipFree(d_output));

  return EXIT_SUCCESS;
}