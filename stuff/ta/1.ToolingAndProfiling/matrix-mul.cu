/*
 * This program executes a matrix multiplication by means of three different implementations:
 * 1) basic algorithm on CPU, 2) basic algorithm on GPU, 3) tiled algorithm on GPU.
 * Analyze the following metrics with nvprof:
 * gld_efficiency,gst_efficiency,gld_transactions,shared_efficiency,shared_load_transactions_per_request,shared_store_transactions_per_request,shared_load_transactions,shared_store_transactions
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
// #include <sys/time.h>

#include <hipblas.h>
#include <iostream>

#define BLOCKDIM 32
#define TILE_WIDTH BLOCKDIM
#define MAXVAL 8

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

using coordinate_type = float;

/*
inline double milliseconds()
{
  struct timeval tp;
  struct timezone tzp;
  int i = gettimeofday(&tp, &tzp);
  return ((double)tp.tv_sec * 1000 + (double)tp.tv_usec * 0.001);
}
*/

// compute matrix multiplication on CPU
void matrixmult(const coordinate_type *__restrict__ M, const coordinate_type *__restrict__ N, coordinate_type *__restrict__ P, const int numMRows, const int numMColumns, const int numNColumns)
{
  for (int i = 0; i < numMRows; i++)
    for (int j = 0; j < numNColumns; j++)
    {
      P[i * numMColumns + j] = {0};
      for (int k = 0; k < numMColumns; k++)
        P[i * numMColumns + j] += M[i * numMColumns + k] * N[k * numNColumns + j];
    }
}

// compute matrix multiplication on GPU
// Naive approach
__global__ void basic_matrixmult(const coordinate_type *__restrict__ M, const coordinate_type *__restrict__ N, coordinate_type *__restrict__ P, const int numMRows, const int numMColumns, const int numNColumns)
{
  const int i = blockIdx.y * blockDim.y + threadIdx.y;
  const int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numMRows && j < numNColumns)
  {
    int sum = 0;
    for (int k = 0; k < numMColumns; k++)
      sum += M[i * numMColumns + k] * N[k * numNColumns + j];
    P[i * numMColumns + j] = sum;
  }
}

// compute tiled matrix multiplication on GPU
// Tiled approach
__global__ void tiled_matrixmult(const coordinate_type *__restrict__ M, const coordinate_type *__restrict__ N, coordinate_type *__restrict__ P, const int numMRows, const int numMColumns, const int numNColumns)
{
  __shared__ coordinate_type ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ coordinate_type ds_N[TILE_WIDTH][TILE_WIDTH];
  const int bx = blockIdx.x,
            by = blockIdx.y,
            tx = threadIdx.x,
            ty = threadIdx.y,
            i = by * TILE_WIDTH + ty,
            j = bx * TILE_WIDTH + tx;
  int Pvalue = 0;

  for (int m = 0; m < (numMColumns - 1) / TILE_WIDTH + 1; ++m)
  {
    if (i < numMRows && m * TILE_WIDTH + tx < numMColumns)
      ds_M[ty][tx] = M[i * numMColumns + m * TILE_WIDTH + tx];
    if (j < numNColumns && m * TILE_WIDTH + ty < numMColumns)
      ds_N[ty][tx] = N[(m * TILE_WIDTH + ty) * numNColumns + j];
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k)
    {
      // NB: this is reading possibly non initialized values,
      // need to add this check here:
      // if (i < numMRows && m * TILE_WIDTH + k < numMColumns && m * TILE_WIDTH + k < numNRows && j < numNColumns)
      // or initialize the shared mem to 0 in the else cases of the 2 above ifs
      Pvalue += ds_M[ty][k] * ds_N[k][tx];
    }
    __syncthreads();
  }
  if (i < numMRows && j < numNColumns)
    P[i * numMColumns + j] = Pvalue;
}

int main(int argc, char **argv)
{
  coordinate_type *h_A;     // The A matrix
  coordinate_type *h_B;     // The B matrix
  coordinate_type *h_C;     // The output C matrix
  coordinate_type *h_C_cpu; // The output C matrix computed on the CPU
  coordinate_type *d_A;
  coordinate_type *d_B;
  coordinate_type *d_C;
  unsigned int numARows;    // number of rows in the matrix A
  unsigned int numAColumns; // number of columns in the matrix A
  unsigned int numBRows;    // number of rows in the matrix B
  unsigned int numBColumns; // number of columns in the matrix B
  unsigned int numCRows;
  unsigned int numCColumns;
  int ok, i;

  if (argc != 5)
  {
    printf("Please specify sizes (#rows and #columns) of matrix A and B\n");
    return 0;
  }
  numARows = atoi(argv[1]);
  numAColumns = atoi(argv[2]);
  numBRows = atoi(argv[3]);
  numBColumns = atoi(argv[4]);

  if (numAColumns != numBRows)
  {
    printf("# colums of A is different from the number of rows of B\n");
    return 0;
  }

  // compute output matrix size
  numCRows = numARows;
  numCColumns = numBColumns;

  // allocate memory for the three matrices
  h_A = (coordinate_type *)malloc(sizeof(coordinate_type) * numARows * numAColumns);
  if (!h_A)
  {
    printf("Error: malloc failed\n");
    return 1;
  }
  h_B = (coordinate_type *)malloc(sizeof(coordinate_type) * numBRows * numBColumns);
  if (!h_B)
  {
    printf("Error: malloc failed\n");
    free(h_A);
    return 1;
  }
  h_C = (coordinate_type *)malloc(sizeof(coordinate_type) * numCRows * numCColumns);
  if (!h_C)
  {
    printf("Error: malloc failed\n");
    free(h_A);
    free(h_B);
    return 1;
  }
  h_C_cpu = (coordinate_type *)malloc(sizeof(coordinate_type) * numCRows * numCColumns);
  if (!h_C_cpu)
  {
    printf("Error: malloc failed\n");
    free(h_A);
    free(h_B);
    free(h_C);
    return 1;
  }
  // initialize input matrices
  srand(0);
  for (i = 0; i < numARows * numAColumns; i++)
    h_A[i] = rand() % MAXVAL;
  for (i = 0; i < numBRows * numBColumns; i++)
    h_B[i] = rand() % MAXVAL;

  // execute on CPU
  matrixmult(h_A, h_B, h_C_cpu, numARows, numAColumns, numBColumns);

  // allocate device memory and transfer data
  CHECK(hipMalloc(&d_A, numARows * numAColumns * sizeof(coordinate_type)));
  CHECK(hipMalloc(&d_B, numBRows * numBColumns * sizeof(coordinate_type)));
  CHECK(hipMalloc(&d_C, numCRows * numCColumns * sizeof(coordinate_type)));
  CHECK(hipMemcpy(d_A, h_A, numARows * numAColumns * sizeof(coordinate_type), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_B, h_B, numBRows * numBColumns * sizeof(coordinate_type), hipMemcpyHostToDevice));

  // execute on GPU 1
  dim3 blockDim(BLOCKDIM, BLOCKDIM);
  dim3 gridDim(ceil(((float)numBColumns) / blockDim.x), ceil(((float)numARows) / blockDim.y));

  basic_matrixmult<<<gridDim, blockDim>>>(d_A, d_B, d_C, numARows, numAColumns, numBColumns);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  // transfer results 1
  CHECK(hipMemcpy(h_C, d_C, numCRows * numCColumns * sizeof(coordinate_type), hipMemcpyDeviceToHost));

  // check results
  for (i = 0, ok = 1; i < numCRows * numCColumns; i++)
    if (h_C[i] != h_C_cpu[i])
    {
      printf("Different numbers %f %f\n", h_C[i], h_C_cpu[i]);
      ok = 0;
    }
  printf("Result: %s\n", ok ? "OK" : "NO");

  // execute 2
  tiled_matrixmult<<<gridDim, blockDim>>>(d_A, d_B, d_C, numARows, numAColumns, numBColumns);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  // transfer results 2
  CHECK(hipMemcpy(h_C, d_C, numCRows * numCColumns * sizeof(coordinate_type), hipMemcpyDeviceToHost));

  // check results
  for (i = 0, ok = 1; i < numCRows * numCColumns; i++)
    if (h_C[i] != h_C_cpu[i])
    {
      printf("Different numbers %f %f\n", h_C[i], h_C_cpu[i]);
      ok = 0;
    }
  printf("Result: %s\n", ok ? "OK" : "NO");

  // execute 3
  hipblasHandle_t handle;
  if (hipblasCreate(&handle))
  {
    std::cerr << "Create cublas handle error." << std::endl;
    exit(EXIT_FAILURE);
  };
  const coordinate_type alpha{1};
  const coordinate_type beta{0};
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, numBColumns, numARows, numAColumns, &alpha, d_B, numBColumns, d_A, numAColumns, &beta, d_C, numBColumns);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());
  hipblasDestroy(handle);

  // transfer results 3
  CHECK(hipMemcpy(h_C, d_C, numCRows * numCColumns * sizeof(coordinate_type), hipMemcpyDeviceToHost));

  // check results
  for (i = 0, ok = 1; i < numCRows * numCColumns; i++)
    if (h_C[i] != h_C_cpu[i])
    {
      printf("Different numbers %f %f\n", h_C[i], h_C_cpu[i]);
      ok = 0;
    }
  printf("Result: %s\n", ok ? "OK" : "NO");

  CHECK(hipFree(d_A));
  CHECK(hipFree(d_B));
  CHECK(hipFree(d_C));

  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_cpu);

  return 0;
}
