#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define BLOCKDIM 32
#define GRIDIM 32
#define GRID_MULTIPLE 1

#define CHECK(call) \
{ \
  const hipError_t err = call; \
  if (err != hipSuccess) { \
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
    exit(EXIT_FAILURE); \
  } \
} \


#define CHECK_KERNELCALL() \
{ \
  const hipError_t err = hipGetLastError();\
  if (err != hipSuccess) {\
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
    exit(EXIT_FAILURE);\
  }\
}\


// Kernel to execute the saxpy
__global__
void saxpy(const int n, const float a, float * __restrict__ x, float * __restrict__ y)
{
  const int tid = blockIdx.x*blockDim.x + threadIdx.x;
  const int stride = blockDim.x*gridDim.x;
  for(int index=tid; index<n; index+=stride)
    y[index] = a*x[index] + y[index];
}

int main(void)
{
  const int N = 1<<28;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  CHECK(hipMalloc(&d_x, N*sizeof(float))); 
  CHECK(hipMalloc(&d_y, N*sizeof(float)));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  CHECK(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice));

  // Insert code to play with GRID's dimensions
  saxpy<<<GRIDIM, BLOCKDIM>>>(N, 2.0f, d_x, d_y);
  CHECK_KERNELCALL();

  CHECK(hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost));

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  CHECK(hipFree(d_x));
  CHECK(hipFree(d_y));
  free(x);
  free(y);
}
