
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include <sys/time.h>
#include <time.h>

#include <iostream>
#include <fstream>
#include <string>

#define MAX_LENGTH 50000000

#define WARPSIZE 32
// For shuffle block dim need to have the same dimension of a warp
#define BLOCKDIM WARPSIZE
#define GRIDDIM 2048

#define CHAR_PER_BIN 6
#define ALPHABET_SIZE 26
#define BIN_NUM ((ALPHABET_SIZE - 1) / CHAR_PER_BIN + 1)
#define FIRST_CHAR 'a'

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess)                                                         \
    {                                                                               \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

/*
double get_time() // function to get the time of day in seconds
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + tv.tv_usec * 1e-6;
}
*/

void sequential_histogram(const char *data, unsigned int *histogram, const int length)
{
  for (int i = 0; i < length; i++)
  {
    int alphabet_position = data[i] - FIRST_CHAR;
    if (alphabet_position >= 0 && alphabet_position < ALPHABET_SIZE) // check if we have an alphabet char
      histogram[alphabet_position / CHAR_PER_BIN]++;                 // we group the letters into blocks of 6
  }
}

__global__ void histogram_kernel(const char *__restrict__ data,
                                 unsigned int *__restrict__ histogram,
                                 const unsigned int length)
{
  const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int stride = blockDim.x * gridDim.x;
  // Privatized bins
  __shared__ unsigned int histo_s[BIN_NUM];
#pragma unroll
  for (unsigned int binIdx = threadIdx.x; binIdx < BIN_NUM; binIdx += blockDim.x)
    histo_s[binIdx] = 0u;
  __syncthreads();
  // Histogram
  for (unsigned int i = tid; i < length; i += stride)
  {
    const int alphabet_position = data[i] - FIRST_CHAR;
    if (alphabet_position >= 0 && alphabet_position < ALPHABET_SIZE)
      atomicAdd(&(histo_s[alphabet_position / CHAR_PER_BIN]), 1);
  }
  __syncthreads();
// Commit to global memory
#pragma unroll
  for (unsigned int binIdx = threadIdx.x; binIdx < BIN_NUM; binIdx += blockDim.x)
  {
    const unsigned int binValue = histo_s[binIdx];
    if (binValue > 0)
    {
      atomicAdd(&(histogram[binIdx]), binValue);
    }
  }
}

int main(int argc, char *argv[])
{
  if (argc < 2)
  {
    printf("Please provide a filename as an argument.\n");
    return 1;
  }

  const char *filename = argv[1];
  std::fstream fp(filename);

  std::string txt;
  while (std::getline(fp, txt))
  {
    printf("Retrieved line of length %zd:\n", txt.length());
  }
  fp.close();

  const char *text = txt.c_str();
  int len = txt.length();
  char *text_d;

  unsigned int histogram[BIN_NUM] = {0};
  unsigned int histogram_hw[BIN_NUM] = {0};
  unsigned int *histogram_d;
  sequential_histogram(text, histogram, len);

  CHECK(hipMalloc(&text_d, len * sizeof(char)));                              // allocate space for the input array on the GPU
  CHECK(hipMalloc(&histogram_d, BIN_NUM * sizeof(unsigned int)));             // and for the histogram
  CHECK(hipMemcpy(text_d, text, len * sizeof(char), hipMemcpyHostToDevice)); // copy input data on the gpu

  dim3 blocksPerGrid(GRIDDIM, 1, 1);
  dim3 threadsPerBlock(BLOCKDIM, 1, 1);
  histogram_kernel<<<blocksPerGrid, threadsPerBlock>>>(text_d,
                                                       histogram_d,
                                                       len); // same nof as the regular histogram
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  CHECK(hipMemcpy(histogram_hw,
                   histogram_d,
                   BIN_NUM * sizeof(unsigned int),
                   hipMemcpyDeviceToHost)); // copy data back from the gpu
  for (size_t i = 0; i < BIN_NUM; i++)
  {
    if (histogram[i] != histogram_hw[i])
    {
      printf("Error on GPU at index: %zd\n", i);
      return 0;
    }
  }
  printf("ALL GPU OK\n");

  CHECK(hipFree(text_d));
  CHECK(hipFree(histogram_d));

  return 1;
}
