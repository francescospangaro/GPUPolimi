/*
 * Vector addition.
 * Version 0: the sum is performed by a function on the CPU
 */

#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define CHECK(call)                                            \
  {                                                            \
    const hipError_t err = (call);                            \
    if (err != hipSuccess)                                    \
    {                                                          \
      printf("%s in %s at line %d\n", hipGetErrorString(err), \
             __FILE__, __LINE__);                              \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  }

#define CHECK_KERNELCALL()                                     \
  {                                                            \
    const hipError_t err = hipGetLastError();                \
    if (err != hipSuccess)                                    \
    {                                                          \
      printf("%s in %s at line %d\n", hipGetErrorString(err), \
             __FILE__, __LINE__);                              \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  }

#define N (1 << 16)
#define BLOCK_SIZE 64

inline double milliseconds()
{
  struct timeval tp;
  struct timezone tzp;
  int i = gettimeofday(&tp, &tzp);
  return ((float)tp.tv_sec * 1.e+3 + (float)tp.tv_usec * 1.e-3);
}

bool arrayEquals(int *a, int *b, int dim)
{
  for (int i = 0; i < dim; ++i)
  {
    if (a[i] != b[i])
    {
      printf("%d %d %d\n", i, a[i], b[i]);
      return false;
    }
  }
  return true;
}

/* CPU function performing vector addition c = a + b */
void vsum(int *a, int *b, int *c, int dim)
{
  int i;
  for (i = 0; i < dim; i++)
    c[i] = a[i] + b[i];
}

/* GPU kernel performing vector addition c = a + b */
__global__ void gpu_vsum(int *a, int *b, int *c, int dim)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < dim)
    c[i] = a[i] + b[i];
}

int main()
{
  int h_va[N], h_vb[N], h_vc_cpu[N], h_vc_gpu[N];

  /* initialize vectors */
  for (int i = 0; i < N; ++i)
  {
    h_va[i] = i;
    h_vb[i] = N - i;
  }

  /* call CPU function */
  double start = milliseconds();
  vsum(h_va, h_vb, h_vc_cpu, N);
  double elapsed_cpu_millis = milliseconds() - start;

  /* allocate GPU vectors */
  int *d_va, *d_vb, *d_vc;
  CHECK(hipMalloc(&d_va, sizeof(h_va)));
  CHECK(hipMalloc(&d_vb, sizeof(h_vb)));
  CHECK(hipMalloc(&d_vc, sizeof(h_vc_gpu)));

  /* create maarker events */
  hipEvent_t startMark, stopMark;
  CHECK(hipEventCreate(&startMark));
  CHECK(hipEventCreate(&stopMark));

  /* Move memory from host to device */
  CHECK(hipMemcpy(d_va, h_va, sizeof(h_va), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_vb, h_vb, sizeof(h_vb), hipMemcpyHostToDevice));

  /* launch kernel */
  CHECK(hipEventRecord(startMark));
  gpu_vsum<<<ceilf(N / (float)BLOCK_SIZE), BLOCK_SIZE>>>(d_va, d_vb, d_vc, N);
  CHECK_KERNELCALL();
  CHECK(hipEventRecord(stopMark));

  /* copy back result */
  CHECK(hipMemcpy(h_vc_gpu, d_vc, sizeof(h_vc_gpu), hipMemcpyDeviceToHost));

  /* calculate elapsed time right after we get synchronized by memcpy */
  float elapsed_gpu_millis;
  CHECK(hipEventElapsedTime(&elapsed_gpu_millis, startMark, stopMark));

  /* we don't print the results... */
  printf("Done! CPU took: %fms, GPU took: %fms\n", elapsed_cpu_millis, elapsed_gpu_millis);
  printf(arrayEquals(h_vc_cpu, h_vc_gpu, N) ? "Same\n" : "Not the same\n");

  /* destroy events */
  hipEventDestroy(startMark);
  hipEventDestroy(stopMark);

  /* free GPU memory */
  hipFree(d_va);
  hipFree(d_vb);
  hipFree(d_vc);

  return 0;
}
