/*
 * The following program elaborates a 2D matrix of integers where each row contains a
 * variable number of elements. In particular, the kernel function receives in input
 * the matrix and sum to each element a value obtained by multiplying the row number
 * by the number of elements in the row.
 * For instance, if we consider the following matrix with 4 rows:
 * 1 2 3 4
 * 1 2
 * 1 1 1 1 1
 * 2
 * The kernel function will compute a new matrix as follows:
 * 1 2 3 4
 * 3 4
 * 11 11 11 11 11
 * 5
 *
 * The described matrix is represented in the program as follows:
 * - An integer array A contains all the elements of the matrix in a linearized way
 * - An integer variable NUMOFELEMS containing the overall number of elements in the matrix
 * - An integer variable ROWS contains the number of rows in the matrix
 * - An integer array COLOFFSETS contains the indexes where the elements of each row starts in A
 * - An integer array COLS contains the length of each row of the matrix
 *
 * Thus, the matrix above is modeled in the program as:
 * A = [1 2 3 4 1 2 1 1 1 1 1 2]
 * NUMOFELEMS = 12
 * ROWS = 4
 * COLOFFSETS = [0, 4, 6, 11]
 * COLS = [4, 2, 5, 1]
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// macros used to generate random data
#define MIN_COLS 5
#define MAX_COLS 10
#define MIN_ROWS 3
#define MAX_ROWS 5
#define MAX_VAL 10
#define BLOCK_DIM 32

void elaborateMatrix(int *a, int rows, int *colOffsets, int *cols, int *b);
void printM(int *a, int rows, int *colOffsets, int *cols);

// kernel functions: sum to each element of the matrix a coefficient obtained
// by multiplying the row number by the row index
void elaborateMatrix(int *a, int rows, int *colOffsets, int *cols, int *b)
{
  int i, j, coeff;
  for (i = 0; i < rows; i++)
  {
    coeff = i * cols[i];
    for (j = 0; j < cols[i]; j++)
      *(b + colOffsets[i] + j) = *(a + colOffsets[i] + j) + coeff;
  }
}

__global__ void elaborateMatrix_gpu(int *a, int rows, int *colOffsets, int *cols, int *b)
{
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;

  if (tx < rows)
  {
    int coeff = tx * cols[tx];
    if (ty < cols[tx])
    {
      *(b + colOffsets[tx] + ty) = *(a + colOffsets[tx] + ty) + coeff;
    }
  }
}

__global__ void elaborateMatrix_child(int *b, int *a, int coeff)
{
  const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  *(b + tid) = *(a + tid) + coeff;
}

__global__ void elaborateMatrix_parent(int *a, int rows, int *colOffsets, int *cols, int *b)
{
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  if (tx < rows)
  {
    int coeff = tx * cols[tx];
    elaborateMatrix_child<<<1, cols[tx]>>>(b + colOffsets[tx], a + colOffsets[tx], coeff);
  }
  __syncthreads();
}



// display the matrix on the screen
void printM(int *a, int rows, int *colOffsets, int *cols)
{
  int i, j;
  for (i = 0; i < rows; i++)
  {
    for (j = 0; j < cols[i]; j++)
      printf("%3d ", *(a + colOffsets[i] + j));
    printf("\n");
  }
}

int main(int argc, char *argv[])
{
  int *a, *b;      // input and output matrices
  int rows;        // number of rows
  int *cols;       // number of columns per each row
  int *colOffsets; // offsets in the data array pointing where each matrix row starts
  int numOfElems;  // overall number of elements
  int i, j;

  // generate the matrix
  srand(0);
  rows = rand() % (MAX_ROWS - MIN_ROWS) + MIN_ROWS;
  cols = (int *)malloc(sizeof(int) * rows);
  if (!cols)
  {
    printf("Error on malloc\n");
    return -1;
  }
  for (i = 0; i < rows; i++)
    cols[i] = rand() % (MAX_COLS - MIN_COLS) + MIN_COLS;

  colOffsets = (int *)malloc(sizeof(int) * rows);
  if (!colOffsets)
  {
    printf("Error on malloc\n");
    return -1;
  }
  for (i = 0, numOfElems = 0; i < rows; i++)
  {
    colOffsets[i] = numOfElems;
    numOfElems += cols[i];
  }

  a = (int *)malloc(sizeof(int) * numOfElems);
  if (!a)
  {
    printf("Error on malloc\n");
    return -1;
  }
  for (i = 0; i < numOfElems; i++)
    a[i] = rand() % MAX_VAL;

  b = (int *)malloc(sizeof(int) * numOfElems);
  if (!b)
  {
    printf("Error on malloc\n");
    return -1;
  }

  // call the kernel function
  elaborateMatrix(a, rows, colOffsets, cols, b);

  // print results
  printM(a, rows, colOffsets, cols);
  printf("\n");
  printM(b, rows, colOffsets, cols);

  int *d_a, *d_b, *d_cols, *d_colOffsets;
  hipMalloc(&d_a, sizeof(int) * numOfElems);
  hipMalloc(&d_b, sizeof(int) * numOfElems);
  hipMalloc(&d_colOffsets, sizeof(int) * rows);
  hipMalloc(&d_cols, sizeof(int) * rows);

  hipMemcpy(d_a, a, sizeof(int) * numOfElems, hipMemcpyHostToDevice);
  hipMemcpy(d_cols, cols, sizeof(int) * rows, hipMemcpyHostToDevice);
  hipMemcpy(d_colOffsets, colOffsets, sizeof(int) * rows, hipMemcpyHostToDevice);

  // dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
  // dim3 blocksPerGrid((rows - 1) / BLOCK_DIM + 1, (MAX_COLS - 1) / BLOCK_DIM + 1);

  // elaborateMatrix_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_a, rows, d_colOffsets, d_cols, d_b);

  dim3 threadsPerBlock(BLOCK_DIM);
  dim3 blocksPerGrid((rows - 1) / BLOCK_DIM + 1);

  elaborateMatrix_parent<<<blocksPerGrid, threadsPerBlock>>>(d_a, rows, d_colOffsets, d_cols, d_b);
  hipDeviceSynchronize();

  int *gpu_res = (int *)malloc(sizeof(int) * numOfElems);
  hipMemcpy(gpu_res, d_b, sizeof(int) * numOfElems, hipMemcpyDeviceToHost);

  printf("\n");
  printM(gpu_res, rows, colOffsets, cols);
  for (i = 0; i < rows; i++)
  {
    for (j = 0; j < cols[i]; j++)
      if (gpu_res[colOffsets[i] + j] != b[colOffsets[i] + j])
      {
        printf("CPU and GPU results are NOT equivalent...\n");
        exit(EXIT_FAILURE);
      }
  }
  printf("ALL OK...\n");

  hipDeviceReset();
  // release memory
  free(a);
  free(b);
  free(cols);
  free(colOffsets);

  return 0;
}