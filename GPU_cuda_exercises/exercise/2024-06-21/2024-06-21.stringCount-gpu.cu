/*
 * This program takes in input a text and a string, being two arrays of char values, and
 * computes how many times the string appears in the text. In particular:
 * - function findString receives in input the text and the string and saves in each
 *   position i of a third vector called match, 1 if an occurrence of the string has been
 *   found in the text starting the index i, 0 otherwise.
 * - function countMatches receives the vector match in input and count the number of values
 *   equal to 1 (i.e., it counts the number of occurrences of the string in the text).
 * - the main function receives as arguments the size of text and string and (for the sake
 *   of brevity) generates randomly the content of the two vectors, invokes the two
 *   functions above and prints the result on the screen.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAXVAL 2
#define BLOCK_DIM 32
#define IN_TILE BLOCK_DIM

void printText(char *text, int num);
void findString(char *text, int textDim, char *str, int strDim, char *match);
int countMatches(char *match, int num);

// display a vector of numbers on the screen
void printText(char *text, int num)
{
  int i;
  for (i = 0; i < num; i++)
    printf("%c", text[i]);
  printf("\n");
}

// kernel function 1: identify strings in the text
void findString(char *text, int textDim, char *str, int strDim, char *match)
{
  int i, j, ok;
  for (i = 0; i < textDim - strDim + 1; i++)
  {
    for (j = 0, ok = 1; j < strDim && ok; j++)
    {
      if (text[i + j] != str[j])
        ok = 0;
    }
    match[i] = ok;
  }
}

__global__ void findString_gpu(char *text, int textDim, char *str, int strDim, char *match)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int ok, j;
  if (tid < textDim - strDim + 1)
  {
    for (j = 0, ok = 1; j < strDim && ok; j++)
    {
      if (text[tid + j] != str[j])
        ok = 0;
    }
    match[tid] = ok;
  }
}

__global__ void findString_shared_gpu(char *text, int textDim, char *str, int strDim, char *match)
{
  extern __shared__ char text_s[];

  const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < textDim)
    text_s[threadIdx.x] = text[tid];
  if (threadIdx.x >= BLOCK_DIM - strDim)
  {
    if (tid + strDim < textDim)
      text_s[threadIdx.x + strDim] = text[tid + strDim];
  }

  __syncthreads();

  int j, ok;
  if (tid < textDim - strDim + 1)
  {
    for (j = 0, ok = 1; j < strDim && ok; j++)
    {
      if (text_s[threadIdx.x + j] != str[j])
        ok = 0;
    }
    match[tid] = ok;
  }
}

// kernel function 2: count matches
int countMatches(char *match, int num)
{
  int i, count;
  for (i = 0, count = 0; i < num; i++)
    count += match[i];
  return count;
}

__global__ void countMatches_gpu(char *match, int *count, int num)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < num)
  {
    atomicAdd(count, match[tid]);
  }
}

int main(int argc, char **argv)
{
  char *text, *str, *match;
  int count;
  int textDim, strDim;
  int i;

  // read arguments
  if (argc != 3)
  {
    printf("Please specify sizes of the two input vectors\n");
    return 0;
  }
  textDim = atoi(argv[1]);
  strDim = atoi(argv[2]);

  // allocate memory for the three vectors
  text = (char *)malloc(sizeof(char) * (textDim));
  if (!text)
  {
    printf("Error: malloc failed\n");
    return 1;
  }
  str = (char *)malloc(sizeof(char) * (strDim));
  if (!str)
  {
    printf("Error: malloc failed\n");
    return 1;
  }
  match = (char *)malloc(sizeof(char) * (textDim - strDim + 1));
  if (!match)
  {
    printf("Error: malloc failed\n");
    return 1;
  }

  // initialize input vectors
  srand(0);
  for (i = 0; i < textDim; i++)
    text[i] = rand() % MAXVAL + 'a';

  for (i = 0; i < strDim; i++)
    str[i] = rand() % MAXVAL + 'a';

  // execute on CPU
  findString(text, textDim, str, strDim, match);
  count = countMatches(match, textDim - strDim + 1);

  // print results
  printf("CPU results:\n");
  printText(text, textDim);
  printText(str, strDim);
  printf("%d\n", count);

  char *d_text, *d_str, *d_match;
  hipMalloc(&d_text, sizeof(char) * textDim);
  hipMalloc(&d_str, sizeof(char) * strDim);
  hipMalloc(&d_match, sizeof(char) * (textDim - strDim + 1));

  hipMemcpy(d_text, text, sizeof(char) * textDim, hipMemcpyHostToDevice);
  hipMemcpy(d_str, str, sizeof(char) * strDim, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(BLOCK_DIM);
  dim3 blocksPerGrid((textDim - strDim) / BLOCK_DIM + 1);

  findString_shared_gpu<<<blocksPerGrid, threadsPerBlock, sizeof(char) * (BLOCK_DIM + strDim)>>>(d_text, textDim, d_str, strDim, d_match);
  hipDeviceSynchronize();

  int *d_count;
  hipMalloc(&d_count, sizeof(int));
  countMatches_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_match, d_count, (textDim - strDim + 1));
  hipDeviceSynchronize();

  int gpu_result;
  hipMemcpy(&gpu_result, d_count, sizeof(int), hipMemcpyDeviceToHost);

  printf("GPU results:\n");
  printText(text, textDim);
  printText(str, strDim);
  printf("%d\n", gpu_result);

  if (gpu_result != count)
  {
    printf("GPU and CPU are NOT equivalent");
    exit(EXIT_FAILURE);
  }

  printf("ALL OK...");
  hipDeviceReset();
  free(text);
  free(str);
  free(match);

  return 0;
}
